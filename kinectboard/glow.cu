#include "hip/hip_runtime.h"
// vim:ts=4:sw=4:expandtab
#include <stdint.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <cutil_inline.h>

#define GRID_X 40
#define GRID_Y 32

#define BLOCK_X 16
#define BLOCK_Y 15

texture<uchar4, 2> gpu_median_masked_tex;
static hipChannelFormatDesc channelDesc;

/*
 * This filter adds a "safety net" of 10 pixels around each pixel which was
 * detected as different in the previous step (the median masking).
 *
 */
__global__ void glow_gpu(uchar4 *gpu_median_masked, uchar4 *gpu_output) {
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int i = (y * 640) + x;

    for (int grow = (y-10); grow < (y+10); grow++) {
        for (int gcol = (x-10); gcol < (x+10); gcol++) {
            /* TODO: check whether it’s within glow_start and glow_end */
            if (tex2D(gpu_median_masked_tex, gcol, grow).x == 255) {
                gpu_output[i].w = 0;
                gpu_output[i].x = 0;
                gpu_output[i].y = 0;
                gpu_output[i].z = 255;
                return;
            }
        }
    }

    gpu_output[i].w = 0;
    gpu_output[i].x = 0;
    gpu_output[i].y = 0;
    gpu_output[i].z = 0;
}

void glow_filter_init(void) {
    channelDesc = hipCreateChannelDesc<uchar4>();
}

void glow_filter(uchar4 *gpu_median_masked, uchar4 *gpu_output) {
    dim3 blocksize(BLOCK_X, BLOCK_Y);
    dim3 gridsize(GRID_X, GRID_Y);

    // XXX: Maybe we could refactor the code so that we don’t need to bind the
    // texture all over again? Nevertheless, the runtime savings are worth
    // doing it.
    cutilSafeCall(hipBindTexture2D(NULL, &gpu_median_masked_tex, gpu_median_masked, &channelDesc, 640, 480, 640 * sizeof(uchar4)));

    glow_gpu<<<gridsize, blocksize>>>(gpu_median_masked, gpu_output);
    if (hipGetLastError() != hipSuccess)
        printf("Could not call kernel. Wrong gridsize/blocksize?\n");

    hipDeviceSynchronize();
    cutilSafeCall(hipUnbindTexture(&gpu_median_masked_tex));
}
